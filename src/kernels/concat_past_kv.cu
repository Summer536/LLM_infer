#include "hip/hip_runtime.h"
// rope之后，k/v shape = [bs, kv_head num, max_q_len, head size] 
// kv cache shape = [num layers, bs, kv_head num, max_seq_len, head size] = >[bs, kv_head num, seqlen[history_len: history_len + max q len] , head size]

#include "src/kernels/concat_past_kv.h"
#include "src/utils/cuda_debug_utils.cuh"
#include <iostream>
template <typename T> 
__global__ void append_key_cache(T *k_dst, // [num layers, bs, kv head num, max_q_len, head size]
                                 const size_t layer_offset,
                                 const T *k_src, // [bs, kv_head num, max_q_len, head size]
                                 const int kv_head_num,
                                 const int head_size,
                                 const int *cur_query_length, 
                                 const int *history_length,   
                                 const int max_q_len,
                                 const int max_seq_len)
{   
    int batch_id = blockIdx.y;   //bs
    int head_id = blockIdx.z;    //head num
    int tid = threadIdx.x;
    int token_id = blockIdx.x;   //sqn_len个Token

    // Now we are at layer_offset, which is the start of the current layer's k cache
    T *k_cache_dst = k_dst + layer_offset;

    int cur_seq_len = cur_query_length[batch_id];
    int cumsum_seq_len = history_length[batch_id]; 
    if (token_id < cur_seq_len) 
    {
        int src_offset = batch_id * kv_head_num * max_q_len * head_size + 
                         head_id * max_q_len * head_size +
                         token_id * head_size + tid;

        int dst_offset = batch_id * kv_head_num * max_seq_len * head_size +
                         head_id * max_seq_len * head_size +
                         (cumsum_seq_len + token_id) * head_size + tid; //写到history_len开始的地方
        k_cache_dst[dst_offset] = k_src[src_offset]; //将K矩阵中的值写入到KV cache中，拼进去
    }
}

template <typename T> 
__global__ void append_value_cache(T *v_dst,
                                   const size_t layer_offset,
                                   const T *v_src,
                                   const int kv_head_num,
                                   const int head_size,
                                   const int *cur_query_length,
                                   const int *history_length,
                                   const int max_q_len,
                                   const int max_seq_len)
{
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int tid = threadIdx.x;
    int token_id = blockIdx.x;

    // Now we are at layer_offset, which is the start of the current layer's v cache
    T *v_cache_dst = v_dst + layer_offset;
    int cur_seq_len = cur_query_length[batch_id];
    int cumsum_seq_len = history_length[batch_id];
    if (token_id < cur_seq_len)
    {
        int src_offset = batch_id * kv_head_num * max_q_len * head_size +
                         head_id * max_q_len * head_size +
                         token_id * head_size + tid;
        int dst_offset = batch_id * kv_head_num * max_seq_len * head_size +
                         head_id * max_seq_len * head_size +
                         (cumsum_seq_len + token_id) * head_size + tid;
        v_cache_dst[dst_offset] = v_src[src_offset];
    }
}

template <typename T>
void launchConcatKVCache(//输入
                         TensorWrapper<T> *k_src, // from qkv bias and rope {batch_size, kv_head_num, max_q_len, head_size}
                         TensorWrapper<T> *v_src,
                         TensorWrapper<int> *layer_id,         // layer offset = layer_id * batchxbeam * max_seq_len * kv_head_num * head_size
                                                            //查询当前的layerid 是layer0 还是layer1 ......
                         TensorWrapper<int> *cur_query_length, // current epoch or local input length,[batchsize]
                                                            //当前输入的句子长度
                         TensorWrapper<int> *history_length,//保存着每个句子的历史上下文长度
                         //输出
                         TensorWrapper<T> *k_dst, //{num_layers, batch_size, kv_head_num, max_seq_len, head_size}
                         TensorWrapper<T> *v_dst) //输出k\v cache
{   
    int batch_size = k_src->shape[0]; 
    int max_seq_len = k_dst->shape[3];
    int kv_head_num = k_src->shape[1];
    int max_q_len = k_src->shape[2];
    int head_size = k_src->shape[3];
    int blockSize = head_size;

    int layer = layer_id->getVal(); 
    size_t layer_offset = layer * batch_size * kv_head_num * max_seq_len * head_size; 

    dim3 grid(max_q_len, batch_size, kv_head_num);
    append_key_cache<T><<<grid, blockSize>>>(k_dst->data,
                                             layer_offset,
                                             k_src->data,
                                             kv_head_num,
                                             head_size,
                                             /*(int*)*/ cur_query_length->data,
                                             /*(int*)*/ history_length->data,
                                             max_q_len,
                                             max_seq_len);

    append_value_cache<T><<<grid, blockSize>>>(v_dst->data,
                                               layer_offset,
                                               v_src->data,
                                               kv_head_num,
                                               head_size,
                                               /*(int*)*/ cur_query_length->data,
                                               /*(int*)*/ history_length->data,
                                               max_q_len,
                                               max_seq_len);

}

template void launchConcatKVCache(TensorWrapper<float> *k_src, // from qkv bias and rope
                                  TensorWrapper<float> *v_src,
                                  TensorWrapper<int> *layer_id,         // layer offset = layer_id * batchxbeam * max_seq_len * kv_head_num * head_size
                                  TensorWrapper<int> *cur_query_length, // current epoch or local input length,[batchsize]
                                  TensorWrapper<int> *history_length,
                                  TensorWrapper<float> *k_dst,
                                  TensorWrapper<float> *v_dst);

template void launchConcatKVCache(TensorWrapper<half> *k_src, // from qkv bias and rope
                                  TensorWrapper<half> *v_src,
                                  TensorWrapper<int> *layer_id,         // layer offset = layer_id * batchxbeam * max_seq_len * kv_head_num * head_size
                                  TensorWrapper<int> *cur_query_length, // current epoch or local input length,[batchsize]
                                  TensorWrapper<int> *history_length,
                                  TensorWrapper<half> *k_dst,
                                  TensorWrapper<half> *v_dst);

template void launchConcatKVCache(TensorWrapper<int8_t> *k_src, // from qkv bias and rope
                                  TensorWrapper<int8_t> *v_src,
                                  TensorWrapper<int> *layer_id,         // layer offset = layer_id * batchxbeam * max_seq_len * kv_head_num * head_size
                                  TensorWrapper<int> *cur_query_length, // current epoch or local input length,[batchsize]
                                  TensorWrapper<int> *history_length,
                                  TensorWrapper<int8_t> *k_dst,
                                  TensorWrapper<int8_t> *v_dst);
