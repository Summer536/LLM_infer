#include "hip/hip_runtime.h"
#include <algorithm> // std::fill_n
#include <iostream>  // snprintf
#include <math.h>    // expf, log
#include <stdlib.h>  // rand
#include <string>    // std::string
#include <vector>    // std::vector

#include <math.h>
#include "src/kernels/attn_softmax_kernel.h"
// there is no cpu kernel implementation now, and if you bought my CUDA lesson, you can find CPU softmax kernel.
// we compare the kernel correctnesss by eyes and result print infos
// `./test_mask_softmax 1` to test half GPU kernel
// `./test_mask_softmax` to test fp32 GPU kernel
#define TEST_MASKED_SOFTMAX(dtype)                                                                                                  \
    dtype *h_qk;                                                                                                                    \
    dtype *d_qk;                                                                                                                    \
    h_qk = (dtype *)malloc(sizeof(dtype) * qk_size);                                                                                \
    hipMalloc((void **)&d_qk, sizeof(dtype) * qk_size);                                                                            \
    dtype *h_score;                                                                                                                 \
    dtype *d_score;                                                                                                                 \
    h_score = (dtype *)malloc(sizeof(dtype) * qk_size);                                                                             \
    hipMalloc((void **)&d_score, sizeof(dtype) * qk_size);                                                                         \
    dtype *h_mask;                                                                                                                \
    dtype *d_mask;                                                                                                                \
    h_mask = (dtype *)malloc(sizeof(dtype) * batch_size * q_length * k_length);                                                 \
    hipMalloc((void **)&d_mask, sizeof(dtype) * batch_size * q_length * k_length);                                               \
    for (int i = 0; i < qk_size; i++)                                                                                               \
    {                                                                                                                               \
        h_qk[i] = i % 8;                                                                                                             \
    }                                                                                                                               \
    for (int i = 0; i < batch_size * q_length * k_length; i++)                                                                      \
    {                                                                                                                               \
        h_mask[i] = (dtype)(1);                                                                                                   \
    }                                                                                                                               \
    hipMemcpy(d_qk, h_qk, sizeof(dtype) * qk_size, hipMemcpyHostToDevice);                                                        \
    hipMemcpy(d_mask, h_mask, sizeof(dtype) * batch_size * q_length * k_length, hipMemcpyHostToDevice);                         \
    DataType type = getTensorType<dtype>();                                                                                         \
    TensorWrapper<dtype> *qk = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, head_num, q_length, k_length}, d_qk);       \
    TensorWrapper<dtype> *mask = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, q_length, k_length}, d_mask);             \
    TensorWrapper<dtype> *score = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, head_num, q_length, k_length}, d_score); \
    std::cout << "before launch softmax kernel" << std::endl;                                                                       \
    launchScaleMaskAndSoftmax(qk, mask, score, scale);                                                                              \
    std::cout << "after launch softmax kernel" << std::endl;                                                                        \
    std::cout << "cuda memcpy device to host" << std::endl;                                                                         \
    hipMemcpy(h_score, score->data, sizeof(dtype) * qk_size, hipMemcpyDeviceToHost);                                              \
    for (int i = 0; i < qk_size; i++)                                                                                               \
    {                                                                                                                               \
        printf("attn score[%d] = %f\n", i, (float)h_score[i]);                                                                      \
    }                                                                                                                               \
    free(h_qk);                                                                                                                     \
    free(h_score);                                                                                                                  \
    free(h_mask);                                                                                                                   \
    hipFree(d_qk);                                                                                                                 \
    hipFree(d_score);                                                                                                              \
    hipFree(d_mask);

// 如果什么都不加直接运行则为测试FP32类型
// ./main 1的时候则为测试FP16类型
int main(int argc, char *argv[])
{
    const int batch_size = 1;
    const int head_num = 2;
    const int q_length = 8;
    const int k_length = 8;
    const int head_size = 4;
    float scale = rsqrtf(float(head_size));
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    const int qk_size = batch_size * head_num * q_length * k_length;
    if (argv[1])
    {
        TEST_MASKED_SOFTMAX(half);  //FP32和FP16的测试代码都是高度一样的，所以这里用了宏定义（之前RMSNorm的UT中，UP写了两份，可以根据这个模板把那个也改为宏，减少代码冗余！）
                                                                                        //(位于/tests/unittests/test_rmsnorm.cu)
    }
    else
    {
        TEST_MASKED_SOFTMAX(float);
    }
}
